#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>
#include <string.h>

#include "z_cu_lshift.cuh"

#define Z_CU_LSHIFT(suffix, type)                                                   \
__global__ void z_cu_lshift_##suffix(z_cu_t* lhs, type rhs, bool* synchro)          \
{                                                                                   \
    size_t const idx = blockIdx.x * blockDim.x + threadIdx.x;                       \
                                                                                    \
    assert(lhs);                                                                    \
                                                                                    \
    z_cu_t* other = 0;                                                              \
    hipMalloc(&other, sizeof(z_cu_t));                                             \
                                                                                    \
    __shared__ bool* synchroTmp;                                                    \
    hipMalloc(&synchroTmp, sizeof(bool));                                          \
                                                                                    \
    SYNCHRO((z_cu_from_##suffix<<<1, 1>>>(other, rhs, synchroTmp)), synchroTmp, 1); \
                                                                                    \
    SYNCHRO((z_cu_lshift_z<<<1, 1>>>(lhs, other, synchroTmp)), synchroTmp, 1);      \
                                                                                    \
    SYNCHRO((z_cu_free<<<1, 1>>>(other, synchroTmp)), synchroTmp, 1);               \
                                                                                    \
    hipFree(synchroTmp);                                                           \
    hipFree(other);                                                                \
                                                                                    \
    synchro[idx] = true;                                                            \
}

Z_CU_LSHIFT(c, char)
Z_CU_LSHIFT(i, int)
Z_CU_LSHIFT(l, long)
Z_CU_LSHIFT(ll, long long)
Z_CU_LSHIFT(s, short)
Z_CU_LSHIFT(uc, unsigned char)
Z_CU_LSHIFT(ui, unsigned int)
Z_CU_LSHIFT(ul, unsigned long)
Z_CU_LSHIFT(ull, unsigned long long)
Z_CU_LSHIFT(us, unsigned short)

__global__ void z_cu_lshift_z(z_cu_t* lhs, z_cu_t const* rhs, bool* synchro)
{
    size_t const idx = blockIdx.x * blockDim.x + threadIdx.x;

    assert(lhs);
    assert(rhs);

    __shared__ bool* synchroTmp;
    hipMalloc(&synchroTmp, sizeof(bool));

    int* cmp1;
    hipMalloc(&cmp1, sizeof(int));

    SYNCHRO((z_cu_cmp_c<<<1, 1>>>(rhs, 0, cmp1, synchroTmp)), synchroTmp, 1);

    assert(*cmp1 >= 0);

    int* cmp2;
    hipMalloc(&cmp2, sizeof(int));

    SYNCHRO((z_cu_cmp_c<<<1, 1>>>(lhs, 0, cmp1, synchroTmp)), synchroTmp, 1);
    SYNCHRO((z_cu_cmp_c<<<1, 1>>>(rhs, 0, cmp2, synchroTmp)), synchroTmp, 1);

    if (!*cmp1 || !*cmp2)
    {
        hipFree(cmp1);
        hipFree(cmp2);
    }
    else if (lhs->is_nan || rhs->is_nan || lhs->is_infinity || rhs->is_infinity)
    {
        hipFree(cmp1);
        hipFree(cmp2);

        SYNCHRO((z_cu_set_nan<<<1, 1>>>(lhs, synchroTmp)), synchroTmp, 1);
    }
    else
    {
        hipFree(cmp1);
        hipFree(cmp2);

        unsigned short const us = sizeof(z_cu_type) * 8;
        z_cu_t* n;
        hipMalloc(&n, sizeof(z_cu_t));
        SYNCHRO((z_cu_div_q_us<<<1, 1>>>(rhs, us, n, synchroTmp)), synchroTmp, 1);

        {
            unsigned long long* ull;
            hipMalloc(&ull, sizeof(unsigned long long));
            SYNCHRO((z_cu_to_ull<<<1, 1>>>(n, ull, synchroTmp)), synchroTmp, 1);

            z_cu_type* bits = 0;
            hipMalloc(&bits, (*ull + lhs->size) * sizeof(z_cu_type));
            assert(bits);
            memset((char*)(bits) + lhs->size * sizeof(z_cu_type), 0, *ull * sizeof(z_cu_type));
            memcpy(bits, lhs->bits, lhs->size * sizeof(z_cu_type));
            hipFree(lhs->bits);
            lhs->bits = bits;
            lhs->size += *ull;

            hipFree(ull);
        }

        z_cu_t* other;
        hipMalloc(&other, sizeof(z_cu_t));
        SYNCHRO((z_cu_copy<<<1, 1>>>(rhs, other, synchroTmp)), synchroTmp, 1);
        z_cu_t* nTmp;
        hipMalloc(&nTmp, sizeof(z_cu_t));
        SYNCHRO((z_cu_copy<<<1, 1>>>(n, nTmp, synchroTmp)), synchroTmp, 1);
        SYNCHRO((z_cu_mul_us<<<1, 1>>>(nTmp, us, synchroTmp)), synchroTmp, 1);
        SYNCHRO((z_cu_sub_z<<<1, 1>>>(other, nTmp, synchroTmp)), synchroTmp, 1);
        SYNCHRO((z_cu_free<<<1, 1>>>(nTmp, synchroTmp)), synchroTmp, 1);
        hipFree(nTmp);

        z_cu_type* bits = 0;
        hipMalloc(&bits, (lhs->size + 1) * sizeof(z_cu_type));
        assert(bits);
        memset(bits, 0, sizeof(z_cu_type));
        memcpy((char*)(bits) + sizeof(z_cu_type), lhs->bits, lhs->size * sizeof(z_cu_type));
        hipFree(lhs->bits);
        lhs->bits = bits;
        ++lhs->size;

        unsigned long long* shift;
        hipMalloc(&shift, sizeof(unsigned long long));
        SYNCHRO((z_cu_to_ull<<<1, 1>>>(other, shift, synchroTmp)), synchroTmp, 1);

        if (*shift)
        {
            for (size_t i = 1; i < lhs->size; ++i)
            {
                longest_type const s = sizeof(z_cu_type) * 8;

                if ((lhs->bits[i] >> (s - *shift)))
                    lhs->bits[i - 1] |= (lhs->bits[i] >> (s - *shift));

                lhs->bits[i] <<= *shift;
            }
        }

        hipFree(shift);

        SYNCHRO((z_cu_free<<<1, 1>>>(n, synchroTmp)), synchroTmp, 1);
        SYNCHRO((z_cu_free<<<1, 1>>>(other, synchroTmp)), synchroTmp, 1);
        hipFree(n);
        hipFree(other);

        if (lhs->is_auto_adjust)
        {
            SYNCHRO((z_cu_adjust<<<1, 1>>>(lhs, synchroTmp)), synchroTmp, 1);
        }
    }

    hipFree(synchroTmp);

    synchro[idx] = true;
}
