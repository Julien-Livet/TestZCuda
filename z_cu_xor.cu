#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdlib.h>
#include <string.h>

#include "z_cu_xor.cuh"

#define Z_CU_XOR(suffix, type)                                                      \
__global__ void z_cu_xor_##suffix(z_cu_t* lhs, type rhs, bool* synchro)             \
{                                                                                   \
    size_t const idx = blockIdx.x * blockDim.x + threadIdx.x;                       \
                                                                                    \
    assert(lhs);                                                                    \
                                                                                    \
    z_cu_t* other = 0;                                                              \
    hipMalloc(&other, sizeof(z_cu_t));                                             \
                                                                                    \
    __shared__ bool* synchroTmp;                                                    \
    hipMalloc(&synchroTmp, sizeof(bool));                                          \
                                                                                    \
    SYNCHRO((z_cu_from_##suffix<<<1, 1>>>(other, rhs, synchroTmp)), synchroTmp, 1); \
                                                                                    \
    SYNCHRO((z_cu_xor_z<<<1, 1>>>(lhs, other, synchroTmp)), synchroTmp, 1);         \
                                                                                    \
    SYNCHRO((z_cu_free<<<1, 1>>>(other, synchroTmp)), synchroTmp, 1);               \
                                                                                    \
    hipFree(synchroTmp);                                                           \
    hipFree(other);                                                                \
                                                                                    \
    synchro[idx] = true;                                                            \
}

Z_CU_XOR(c, char)
Z_CU_XOR(i, int)
Z_CU_XOR(l, long)
Z_CU_XOR(ll, long long)
Z_CU_XOR(s, short)
Z_CU_XOR(uc, unsigned char)
Z_CU_XOR(ui, unsigned int)
Z_CU_XOR(ul, unsigned long)
Z_CU_XOR(ull, unsigned long long)
Z_CU_XOR(us, unsigned short)

__global__ void z_cu_xor_z(z_cu_t* lhs, z_cu_t const* rhs, bool* synchro)
{
    size_t const idx = blockIdx.x * blockDim.x + threadIdx.x;

    assert(lhs);
    assert(rhs);

    if (lhs->size < rhs->size)
    {
        z_cu_type* bits = lhs->bits;
        lhs->bits = 0;
        hipMalloc(&lhs->bits, rhs->size * sizeof(z_cu_type));
        assert(lhs->bits);
        memset(lhs->bits, 0, (rhs->size - lhs->size) * sizeof(z_cu_type));
        memcpy((char*)(lhs->bits) + (rhs->size - lhs->size) * sizeof(z_cu_type), bits, lhs->size * sizeof(z_cu_type));
        lhs->size = rhs->size;
        hipFree(bits);
    }
    else if (lhs->size > rhs->size)
    {
        for (size_t i = 0; i < lhs->size - rhs->size; ++i)
            lhs->bits[i] ^= 0;
    }

    for (size_t i = 0; i < MIN(lhs->size, rhs->size); ++i)
        lhs->bits[lhs->size - 1 - i] ^= rhs->bits[rhs->size - 1 - i];



    if (lhs->is_auto_adjust)
    {
        __shared__ bool* synchroTmp;
        hipMalloc(&synchroTmp, sizeof(bool));

        SYNCHRO((z_cu_adjust<<<1, 1>>>(lhs, synchroTmp)), synchroTmp, 1);

        hipFree(synchroTmp);
    }

    synchro[idx] = true;
}
